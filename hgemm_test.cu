#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <chrono>

#define INDEX(i,j,ld) (((j)*(ld))+(i))

// Uncomment below to print debug statements
// #define DEBUG

using namespace std;


int hgemm_test(int M, int N) {

    
    int i, j;              // Array index iterators
    
    hipError_t error;
    
    hipblasStatus_t status;
    hipblasHandle_t handle;
    
    float* arrayA = 0;     // Will store a copy of the array on the host device
    float* arrayB = 0;     // Will store a copy of the array on the host device
    float* arrayC = 0;     // Will store a copy of the array on the host device

    float* deviceArrayA;   // Will store a copy of the array on the device
    float* deviceArrayB;   // Will store a copy of the array on the device
    float* deviceArrayC;   // Will store a copy of the array on the device

    arrayA = (float *)malloc(M * N * sizeof(arrayA));
    arrayB = (float *)malloc(M * N * sizeof(arrayB));
    arrayC = (float *)malloc(M * N * sizeof(arrayC));
    
    // Ensures that host memory is allocated.
    if (!arrayA || !arrayB || !arrayC) {
        printf("Host memory allocation failed\n");
        return -1;
    }
    
    // Fills all Host Arrays with values
    for (j = 0; j < N; j++) {
        // Iterates Through Rows
        for (i = 0; i < M; i++) {
            // Iterates Through Columns
            arrayA[INDEX(i,j,M)] = (float)(i * M + j + 1);
            arrayB[INDEX(i,j,M)] = (float)(i * M + j + 2);
            arrayC[INDEX(i,j,M)] = 0;
        }
    }
    
    error = hipMalloc((void**)&deviceArrayA, M*N*sizeof(*arrayA));
    if (error != hipSuccess) {
        printf("Device Memory Allocation Failed\n");
        free(arrayA);
        free(arrayB);
        free(arrayC);
        return -1;
    }    

    
    error = hipMalloc((void**)&deviceArrayB, M*N*sizeof(*arrayB));
    if (error != hipSuccess) {
        printf("Device Memory Allocation Failed\n");
        hipFree(deviceArrayA);
        free(arrayA);
        free(arrayB);
        free(arrayC);
        return -1;
    }
    
    error = hipMalloc((void**)&deviceArrayC, M*N*sizeof(*arrayC));
    if (error != hipSuccess) {
        printf("Device Memory Allocation Failed\n");
        hipFree(deviceArrayA);
        hipFree(deviceArrayB);
        free(arrayA);
        free(arrayB);
        free(arrayC);
        return -1;
    }
    
    #ifdef DEBUG
        printf("Device Memory Allocated Successfully\n");
    #endif    

    // Initialize CUBLAS API
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API Initialization Failed\n");
        hipFree(deviceArrayA);
        hipFree(deviceArrayB);
        hipFree(deviceArrayC);
        free(arrayA);
        free(arrayB);
        free(arrayC);
        return -1;
    }
    
    #ifdef DEBUG
        printf("cuBLAS API Initialized Successfully\n");
    #endif
    
    // Move Matrix A from Host to Device
    // NOTE: M = Row count & Leading Dimension A/B(lda/ldb)
    // NOTE: N = Col count
    status = hipblasSetMatrix(M, N, sizeof(*arrayA), arrayA, M, deviceArrayA, M);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR: Failed to move data to device.\n");    
        printf("Freeing Memory and exiting\n");
        hipFree(deviceArrayA);
        hipFree(deviceArrayB);
        hipFree(deviceArrayC);
        free(arrayA);
        free(arrayB);
        free(arrayC);
        hipblasDestroy(handle);
        return -1;
    }
    
    // Move Matrix B from Host to Device
    // NOTE: M = Row count & Leading Dimension A/B(lda/ldb)
    // NOTE: N = Col count
    status = hipblasSetMatrix(M, N, sizeof(*arrayB), arrayB, M, deviceArrayB, M);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR: Failed to move data to device.\n");
        printf("Freeing Memory and exiting\n");
        hipFree(deviceArrayA);
        hipFree(deviceArrayB);
        hipFree(deviceArrayC);

        free(arrayA);
        free(arrayB);

        free(arrayC);
        hipblasDestroy(handle);
        return -1;
    }
    
    
    // Move Matrix B from Host to Device
    // NOTE: M = Row count & Leading Dimension A/B(lda/ldb)
    // NOTE: N = Col count
    status = hipblasSetMatrix(M, N, sizeof(*arrayC), arrayC, M, deviceArrayC, M);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR: Failed to move data to device.\n");
        printf("Freeing Memory and exiting\n");
        hipFree(deviceArrayA);
        hipFree(deviceArrayB);
        hipFree(deviceArrayC)
;
        free(arrayA);
        free(arrayB);
        free(arrayC);
        hipblasDestroy(handle);
        return -1;
    }
    
    #ifdef DEBUG
        printf("SUCCESS: Data transfered to device\n");
    #endif
    
    // Currently set to 1 and 0 respectively to test GEMM functionality
    float alphaScalar = 1.0f;
    float betaScalar = 0.0f;
    
    
    hipblasOperation_t transA = HIPBLAS_OP_N;
    hipblasOperation_t transB = HIPBLAS_OP_N;
    
    #ifdef DEBUG
        printf("Performing GPU Operation\n");
    #endif
    
    // Marks current time for measuring performance
    auto start = chrono::high_resolution_clock::now();
    
    cublasSgemmEx(handle, 
                  transA, 
                  transB, 
                  M, 
                  N, 
                  N, 
                  &alphaScalar, 
                  deviceArrayA, 
                  HIP_R_16F, 
                  M, 
                  deviceArrayB, 
                  HIP_R_16F, 
                  M, 
                  &betaScalar, 
                  deviceArrayC, 
                  HIP_R_16F, 
                  M);
    
    
    // Forces device to finish before proceeding (for timing)
    hipDeviceSynchronize();

    
    // Marks current time for measuring performance
    auto finish = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed = finish - start;
    
    printf("HGEMM: %d x %d -- %fs\n",M,N,elapsed.count());
    
    
    /*

    cublasHgemm(handle, transA, transB, M, N, N, 
                &alphaScalar, deviceArrayA, M, 
                deviceArrayB, M, &betaScalar, 
                deviceArrayC, M);
    */
    
    status = hipblasGetMatrix (M, N, sizeof(*arrayC), deviceArrayC, M, arrayC, M);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data Download Failed\n");
        hipFree(deviceArrayA);
        hipFree(deviceArrayB);
        hipFree(deviceArrayC);
        free(arrayA);
        free(arrayB);    
        free(arrayC);
        hipblasDestroy(handle);
        return -1;
    }
    
    #ifdef DEBUG
        printf("GPU Operation Completed\nFreeing Resources\n");
    #endif
    
    // Frees device pointers from cuda memory
    hipFree(deviceArrayA);
    hipFree(deviceArrayB);
    hipFree(deviceArrayC);
    
    // Uninitializes the cuBLAS hanlder
    hipblasDestroy(handle);

    // Frees host pointers from memory
    free(arrayA);
    free(arrayB);
    free(arrayC);
    
    return EXIT_SUCCESS;

}

int main() {
    
    for (int x = 0; x <= 30000; x += 5000) {
        if (x != 0) {
            hgemm_test(x,x);
        }
    }
    
    return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

using namespace std;


__global__
void add(int n, float *x, float *y) {
    //contains the index of the current thread within its block
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //contins the number of threads in the block
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}


//I don't relly know what to call this function so this describes its
//functionality and it will be named add2 because its basically add + extra stuff
//z = alpha*x + beta* y
//where z, x, and y are vectors of length N, and alpha and beta are scalars.
__global__
void add2(int n, float *x, float *y, float a, float b) {
    float *z = new float[n];

    //contains the index of the current thread within its block
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //contains the number of threads in the block
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        z[i] = (a * x[i]) + (b * y[i]);
    }
}


/*  Ok so I am still figuring this out but here is my understanding
    of the following function:
    NOTE: This code is modified from an example provided by nVidia

    1) GPU Kernel is passed mem address of matrix/vector to analyze.
    2) GPU Kernel is also passed mem address of where to store final result.
    3) Current thread_id and stride is set.
    4) 


*/
__global__ void verticalOperation(int *global_input_data, int *global_output_data) {
    extern __shared__ int shared_data[];
    //each thread loads one element from global memory into shared memory
    int thread_id = threadIdx.x;
    int stride = blockIdx.x * blockDim.x + threadIdx.x;
    shared_data[thread_id] = global_input_data[stride];
    __syncthreads();
    //do reduction in shared memory
    for(int s=1; s < blockDim.x; s *= 2) {
        if (thread_id % (2*s) == 0) {
        shared_data[thread_id] += shared_data[thread_id + s];
    }
    __syncthreads();
}
    // write result for this block to global mem
    if (thread_id == 0) {
        global_output_data[blockIdx.x] = shared_data[0];
    }
}

int main() {

    //For my own sanity lets explain this.
    //1<<20 is a notation that in this context represents
    //a bitshift. That means that you have the bit 1 and then you shift it to the
    //(in this case) left by 20 spaces and fill the empty space with zeros.
    int N = 1<<30; // 1M elements

    float *x, *y;

    //Allocates "Unified Memory" which is accessible from both the CPU and GPU.
    hipError_t cudaMallocErr1 = hipMallocManaged(&x, N*sizeof(float));
    if (cudaMallocErr1 != hipSuccess) {
        cout << "CUDA Error" << endl;
    }
    hipError_t cudaMallocErr2 = hipMallocManaged(&y, N*sizeof(float));
    if (cudaMallocErr2 != hipSuccess) {
        cout << "CUDA Error" << endl;
    }

    //initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    //Runs cuda kernel on 1M elements on the CPU
    int blockSize = 256;
    int numBlocks = (N + blockSize -1) / blockSize;

    add<<<numBlocks, blockSize>>>(N, x, y);
    add2<<<numBlocks, blockSize>>>(N, x, y, 4.0, 5.0);

    cout << verticalOperation<<<numBlocks, blockSize>>>(N, x);

    cout << "Done!" << endl;

    //Forces CPU to wait for GPU to finish before accessing
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;

}

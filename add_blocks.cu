
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;


__global__
void add(int n, float *x, float *y) {
    //contains the index of the current thread within its block
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //contins the number of threads in the block
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}


//I don't relly know what to call this function so this describes its
//functionality and it will be named add2 because its basically add + extra stuff
//z = alpha*x + beta* y
//where z, x, and y are vectors of length N, and alpha and beta are scalars.
__global__
void add2(int n, float *x, float *y, float a, float b) {
     float *z = new float[n];
     for (int i = 0; i < n; i++) {
         z[i] = (a * x[i]) + (b * y[i]);
     }
}

int main() {

    //For my own sanity lets explain this.
    //1<<20 is a notation that in this context represents
    //a bitshift. That means that you have the bit 1 and then you shift it to the
    //(in this case) left by 20 spaces and fill the empty space with zeros.
    int N = 1<<30; // 1M elements

    float *x, *y;

    //Allocates "Unified Memory" which is accessible from both the CPU and GPU.
    hipError_t cudaMallocErr1 = hipMallocManaged(&x, N*sizeof(float));
    if (cudaMallocErr1 != hipSuccess) {
        cout << "CUDA Error" << endl;
    }
    hipError_t cudaMallocErr2 = hipMallocManaged(&y, N*sizeof(float));
    if (cudaMallocErr2 != hipSuccess) {
        cout << "CUDA Error" << endl;
    }

    //initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    //Runs cuda kernel on 1M elements on the CPU
    int blockSize = 256;
    int numBlocks = (N + blockSize -1) / blockSize;

    add<<<numBlocks, blockSize>>>(N, x, y);
    add2<<<numBlocks, blockSize>>>(N, x, y, 4.0, 5.0);


    cout << "Done!" << endl;

    //Forces CPU to wait for GPU to finish before accessing
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;

} 

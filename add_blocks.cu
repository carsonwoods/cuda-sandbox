
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;


__global__
void add(int n, float *x, float *y) {
    //contains the index of the current thread within its block
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //contins the number of threads in the block
    int stride = blockDim.x * gridDim.x;



    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int main() {

    //For my own sanity lets explain this.
    //1<<20 is a notation that in this context represents
    //a bitshift. That means that you have the bit 1 and then you shift it to the
    //(in this case) left by 20 spaces and fill the empty space with zeros.
    int N = 1<<25; // 1M elements

    float *x, *y;

    //Allocates "Unified Memory" which is accessible from both the CPU and GPU.
    hipError_t err = hipMallocManaged(&x, N*sizeof(float));
    if (err != hipSuccess) {
	cout << "CUDA Error" << endl;
   	//printf("%s\n", cudaGetErrorString(err));
    }	 
   // cudaMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    //initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    //Runs cuda kernel on 1M elements on the CPU
    int blockSize = 256;
    int numBlocks = (N + blockSize -1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    cout << "Add Completed" << endl;

    //Forces CPU to wait for GPU to finish before accessing
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
} 

#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

using namespace std;

__global__ void add(float *dX, float *dY) {

    // contains the index of the current thread in the block
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    

    valuesPerThread = 1;


    // Each threads will iterate through all assigned values
    for (int i = 0; i < valuesPerThread; i++) {
        dY[index+i] = dX[index+i] + dY[index+i];
    }
    
    
}


int main() {
    
    int N = 256;
    int memSize = N*sizeof(float);
    
    float x[N], y[N], z[N];
    
    float *dX, *dY;
    
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 50.0f;
    }
    
    hipMalloc(&dX, memSize);
    hipMalloc(&dY, memSize);
    
    hipMemcpy(dX, &x, memSize, hipMemcpyHostToDevice);
    hipMemcpy(dY, &y, memSize, hipMemcpyHostToDevice);

    
    
    add<<<1, 50>>>(dX, dY);
    
    hipDeviceSynchronize();
    
    hipMemcpy(&z, dY, memSize, hipMemcpyDeviceToHost);
    
    hipFree(dX);
    hipFree(dY);

    printf("Done!\n");
    printf("Memory freed\n");
    
    return 0;
    
}
